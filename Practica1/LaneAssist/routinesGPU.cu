#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

#include "routinesGPU.h"
#define DEG2RAD 0.017453f
#define BLOCKSZ 16
#define PADDING 1


__global__ void NRcanny(uint8_t *im, float *NR, int height, int width){

	unsigned int i = (blockIdx.y * blockDim.y) + threadIdx.y, j = (blockIdx.x * blockDim.x) + threadIdx.x;
	unsigned int ii = (threadIdx.y) + 2, jj = (threadIdx.x) + 2;

	__shared__ uint8_t sh_im[2 + BLOCKSZ + 2 + PADDING][2 + BLOCKSZ + 2];

	if(i < height && j < width){
		sh_im[jj][ii] = im[(i * width) + j];
	}


	if((ii == 2 || ii == 3) && i >= 2){ 
		sh_im[jj][ii - 2] = im[((i - 2) * width) + j];
		if((jj == 2 || jj == 3) && j >= 2){
			sh_im[jj - 2][ii - 2] = im[((i - 2) * width) + (j - 2)]; 
		}
	}
	if((jj == 2 || jj == 3) && j >= 2){
		sh_im[jj - 2][ii] = im[(i * width) + (j - 2)];
		if((ii == BLOCKSZ + 1 || ii == BLOCKSZ) && i < (height - 2)){
			sh_im[jj - 2][ii + 2] = im[((i + 2) * width) + (j - 2)];
		}
	}
	if((ii == BLOCKSZ + 1 || ii == BLOCKSZ) && i < (height - 2)){ 
		sh_im[jj][ii + 2] = im[((i + 2) * width) + j];
		if((jj == BLOCKSZ + 1 || jj == BLOCKSZ) && j < (width - 2)){
			sh_im[jj + 2][ii + 2] = im[((i + 2) * width) + (j + 2)];
		}
	}
	if((jj == BLOCKSZ + 1 || jj == BLOCKSZ) && j < (width - 2)){
		sh_im[jj + 2][ii] = im[(i * width) + (j + 2)];
		if((ii == 2 ||ii == 3) && i >= 2){
			sh_im[jj + 2][ii - 2] = im[((i - 2) * width) + (j + 2)]; 
		}
	}

	__syncthreads();

	if(i >= 2 && j >= 2 && i < (height - 2) && j < (width - 2)){
		// Noise reduction
		NR[(i * width) + j] = 
			 (2.0*sh_im[jj - 2][ii - 2] +  4.0*sh_im[jj - 1][ii - 2] +  5.0*sh_im[jj    ][ii - 2] +  4.0*sh_im[jj + 1][ii - 2] + 2.0*sh_im[jj + 2][ii - 2]
			+ 4.0*sh_im[jj - 2][ii - 1] +  9.0*sh_im[jj - 1][ii - 1] + 12.0*sh_im[jj    ][ii - 1] +  9.0*sh_im[jj + 1][ii - 1] + 4.0*sh_im[jj + 2][ii - 1]
			+ 5.0*sh_im[jj - 2][ii    ] + 12.0*sh_im[jj - 1][ii    ] + 15.0*sh_im[jj    ][ii    ] + 12.0*sh_im[jj + 1][ii    ] + 5.0*sh_im[jj + 2][ii    ]
			+ 4.0*sh_im[jj - 2][ii + 1] +  9.0*sh_im[jj - 1][ii + 1] + 12.0*sh_im[jj    ][ii + 1] +  9.0*sh_im[jj + 1][ii + 1] + 4.0*sh_im[jj + 2][ii + 1]
			+ 2.0*sh_im[jj - 2][ii + 2] +  4.0*sh_im[jj - 1][ii + 2] +  5.0*sh_im[jj    ][ii + 2] +  4.0*sh_im[jj + 1][ii + 2] + 2.0*sh_im[jj + 2][ii + 2])
			/159.0;
			
	}
}
__global__ void Gcanny(float *G, float *NR, float *phi, int height, int width){
	
	unsigned int i = (blockIdx.y * blockDim.y) + threadIdx.y, j = (blockIdx.x * blockDim.x) + threadIdx.x;
	unsigned int ii = (threadIdx.y) + 2, jj = (threadIdx.x) + 2;

	__shared__ float sh_NR[2 + BLOCKSZ + 2 + PADDING][2 + BLOCKSZ + 2];

	float Gy, Gx, phi_thread;
	float PI = 3.141593;

	if(i < height && j < width){
		sh_NR[jj][ii] = NR[(i * width) + j];
	}


	if((ii == 2 || ii == 3) && i >= 2){ 
		sh_NR[jj][ii - 2] = NR[((i - 2) * width) + j];
		if((jj == 2 || jj == 3) && j >= 2){
			sh_NR[jj - 2][ii - 2] = NR[((i - 2) * width) + (j - 2)]; 
		}
	}
	if((jj == 2 || jj == 3) && j >= 2){
		sh_NR[jj - 2][ii] = NR[(i * width) + (j - 2)];
		if((ii == BLOCKSZ + 1 || ii == BLOCKSZ) && i < (height - 2)){
			sh_NR[jj - 2][ii + 2] = NR[((i + 2) * width) + (j - 2)];
		}
	}
	if((ii == BLOCKSZ + 1 || ii == BLOCKSZ) && i < (height - 2)){ 
		sh_NR[jj][ii + 2] = NR[((i + 2) * width) + j];
		if((jj == BLOCKSZ + 1 || jj == BLOCKSZ) && j < (width - 2)){
			sh_NR[jj + 2][ii + 2] = NR[((i + 2) * width) + (j + 2)];
		}
	}
	if((jj == BLOCKSZ + 1 || jj == BLOCKSZ) && j < (width - 2)){
		sh_NR[jj + 2][ii] = NR[(i * width) + (j + 2)];
		if((ii == 2 ||ii == 3) && i >= 2){
			sh_NR[jj + 2][ii - 2] = NR[((i - 2) * width) + (j + 2)]; 
		}
	}

	__syncthreads();

	if(i >= 2 && j >= 2 && i < (height - 2) && j < (width - 2)){
		// Intensity gradient of the image
		Gx = 
			 (1.0*sh_NR[jj - 2][ii - 2] +  2.0*sh_NR[jj - 1][ii - 2] +  (-2.0)*sh_NR[jj + 1][ii - 2] + (-1.0)*sh_NR[jj + 2][ii - 2]
	   		+ 4.0*sh_NR[jj - 2][ii - 1] +  8.0*sh_NR[jj - 1][ii - 1] +  (-8.0)*sh_NR[jj + 1][ii - 1] + (-4.0)*sh_NR[jj + 2][ii - 1]
	   		+ 6.0*sh_NR[jj - 2][ii    ] + 12.0*sh_NR[jj - 1][ii    ] + (-12.0)*sh_NR[jj + 1][ii    ] + (-6.0)*sh_NR[jj + 2][ii    ]
	   		+ 4.0*sh_NR[jj - 2][ii + 1] +  8.0*sh_NR[jj - 1][ii + 1] +  (-8.0)*sh_NR[jj + 1][ii + 1] + (-4.0)*sh_NR[jj + 2][ii + 1]
	  		+ 1.0*sh_NR[jj - 2][ii + 2] +  2.0*sh_NR[jj - 1][ii + 2] +  (-2.0)*sh_NR[jj + 1][ii + 2] + (-1.0)*sh_NR[jj + 2][ii + 2]);


   		Gy = 
			((-1.0)*sh_NR[jj - 2][ii - 2] + (-4.0)*sh_NR[jj - 1][ii - 2] +  (-6.0)*sh_NR[jj    ][ii - 2] + (-4.0)*sh_NR[jj + 1][ii - 2] + (-1.0)*sh_NR[jj + 2][ii - 2]
	   		+(-2.0)*sh_NR[jj - 2][ii - 1] + (-8.0)*sh_NR[jj - 1][ii - 1] + (-12.0)*sh_NR[jj    ][ii - 1] + (-8.0)*sh_NR[jj + 1][ii - 1] + (-2.0)*sh_NR[jj + 2][ii - 1]
	   		+   2.0*sh_NR[jj - 2][ii + 1] +    8.0*sh_NR[jj - 1][ii + 1] +    12.0*sh_NR[jj    ][ii + 1] +    8.0*sh_NR[jj + 1][ii + 1] +    2.0*sh_NR[jj + 2][ii + 1]
	   		+   1.0*sh_NR[jj - 2][ii + 2] +    4.0*sh_NR[jj - 1][ii + 2] +     6.0*sh_NR[jj    ][ii + 2] +    4.0*sh_NR[jj + 1][ii + 2] +    1.0*sh_NR[jj + 2][ii + 2]);
		
		G[i*width+j] = sqrtf((Gx*Gx)+(Gy*Gy));	//G = √Gx²+Gy²
		phi_thread = atan2f(fabs(Gy),fabs(Gx));

		if(fabs(phi_thread)<=PI/8 ) phi[i*width+j] = 0;
		else if (fabs(phi_thread)<= 3*(PI/8)) phi[i*width+j] = 45;
		else if (fabs(phi_thread) <= 5*(PI/8)) phi[i*width+j] = 90;
		else if (fabs(phi_thread) <= 7*(PI/8)) phi[i*width+j] = 135;
		else phi[i*width+j] = 0;
	}

	
}
__global__ void PEDGEcanny(float *G, uint8_t *imEdge, float *phi, int height, int width, float level){

	unsigned int i = (blockIdx.y * blockDim.y) + threadIdx.y, j = (blockIdx.x * blockDim.x) + threadIdx.x;
	unsigned int ii = (threadIdx.y) + 3, jj = (threadIdx.x) + 3;
	int iii = 0, jjj = 0;

	__shared__ float sh_G[3 + BLOCKSZ + 3 + PADDING][3 + BLOCKSZ + 3];

	float phi_thread, G_thread, lowthres = (level/2), hithres = 2*(level);
	uint8_t pedge = 0;

	if(i < height && j < width){
		sh_G[jj][ii] = G[(i * width) + j];
	}

	if(ii == 3 && i >= 3){
		sh_G[jj][ii - 1] = G[((i - 1) * width) + j]; 
		if(jj == 3 && j >= 3){
			sh_G[jj - 1][ii - 1] = G[((i - 1) * width) + (j - 1)];
		}
	}
	if(jj == 3 && j >= 3){
		sh_G[jj - 1][ii] = G[(i * width) + (j - 1)]; 
		if(ii == BLOCKSZ + 2 && i < (height - 3)){
			sh_G[jj - 1][ii + 1] = G[((i + 1) * width) + (j - 1)]; 

		}
	}
	if(ii == BLOCKSZ + 2 && i < (height - 3)){
		sh_G[jj][ii + 1] = G[((i + 1) * width) + j]; 
		if(jj == BLOCKSZ + 2 && j < (width - 3)){
			sh_G[jj + 1][ii + 1] = G[((i + 1) * width) + (j + 1)]; 
		}
	}
	if(jj == BLOCKSZ + 2 && j < (width - 3)){
		sh_G[jj + 1][ii] = G[(i * width) + (j + 1)]; 
		if(ii == 3 && i >= 3){
			sh_G[jj + 1][ii - 1] = G[((i - 1) * width) + (j + 1)];
		}
	}

	__syncthreads();

	if(i >= 3 && j >= 3 && i < (height - 3) && j < (width - 3)){
		phi_thread = phi[i*width+j];
		G_thread = sh_G[jj][ii];
		if(phi_thread == 0){
			if(G_thread > sh_G[jj + 1][ii] && G_thread > sh_G[jj - 1][ii]) //edge is in N-S
			pedge = 1;

		} else if(phi_thread == 45) {
			if(G_thread > sh_G[jj + 1][ii + 1] && G_thread > sh_G[jj - 1][ii - 1]) // edge is in NW-SE
			pedge= 1;

		} else if(phi_thread == 90) {
			if(G_thread > sh_G[jj][ii + 1] && G_thread > sh_G[jj][ii - 1]) //edge is in E-W
			pedge = 1;

		} else if(phi_thread == 135) {
			if(G_thread > sh_G[jj - 1][ii + 1] && G_thread > sh_G[jj + 1][ii - 1]) // edge is in NE-SW
			pedge = 1;
		}

		if(G_thread > hithres && pedge ){ imEdge[i*width+j] = 255; }
		else if(pedge && G_thread >= lowthres && G_thread < hithres){
			// check neighbours 3x3
			for (iii = -1; iii <= 1; iii++){
				for (jjj = -1; jjj <= 1; jjj++){
					if (sh_G[jj + iii][ii+jjj] > hithres) {imEdge[i*width+j] = 255; iii = 2; jjj = 2;}
				}
			}
		}
	}

}


void getlines(int threshold, uint32_t *accumulators, int accu_width, int accu_height, int width, int height, 
	float *sin_table, float *cos_table,
	int *x1_lines, int *y1_lines, int *x2_lines, int *y2_lines, int *lines)
{
	int rho, theta, ii, jj;
	uint32_t max;

	for(rho=0;rho<accu_height;rho++)
	{
		for(theta=0;theta<accu_width;theta++)  
		{  

			if(accumulators[(rho*accu_width) + theta] >= threshold)  
			{  
				//Is this point a local maxima (9x9)  
				max = accumulators[(rho*accu_width) + theta]; 
				for(int ii=-4;ii<=4;ii++)  
				{  
					for(int jj=-4;jj<=4;jj++)  
					{  
						if( (ii+rho>=0 && ii+rho<accu_height) && (jj+theta>=0 && jj+theta<accu_width) )  
						{  
							if( accumulators[((rho+ii)*accu_width) + (theta+jj)] > max )  
							{
								max = accumulators[((rho+ii)*accu_width) + (theta+jj)];
							}  
						}  
					}  
				}  
				//local maxima
				if(max == accumulators[(rho*accu_width) + theta]){
					int x1, y1, x2, y2;  
					x1 = y1 = x2 = y2 = 0;  

					if(theta >= 45 && theta <= 135)  
					{
						if (theta>90) {
							//y = (r - x cos(t)) / sin(t)  
							x1 = width/2;  
							y1 = ((float)(rho-(accu_height/2)) - ((x1 - (width/2) ) * cos_table[theta])) / sin_table[theta] + (height / 2);
							x2 = width;  
							y2 = ((float)(rho-(accu_height/2)) - ((x2 - (width/2) ) * cos_table[theta])) / sin_table[theta] + (height / 2);  
						} else {
							//y = (r - x cos(t)) / sin(t)  
							x1 = 0;  
							y1 = ((float)(rho-(accu_height/2)) - ((x1 - (width/2) ) * cos_table[theta])) / sin_table[theta] + (height / 2);
							x2 = width*2/5;  
							y2 = ((float)(rho-(accu_height/2)) - ((x2 - (width/2) ) * cos_table[theta])) / sin_table[theta] + (height / 2); 
						}
					} else {
						//x = (r - y sin(t)) / cos(t);  
						y1 = 0;  
						x1 = ((float)(rho-(accu_height/2)) - ((y1 - (height/2) ) * sin_table[theta])) / cos_table[theta] + (width / 2);  
						y2 = height;  
						x2 = ((float)(rho-(accu_height/2)) - ((y2 - (height/2) ) * sin_table[theta])) / cos_table[theta] + (width / 2);  
					}
					x1_lines[*lines] = x1;
					y1_lines[*lines] = y1;
					x2_lines[*lines] = x2;
					y2_lines[*lines] = y2;
					(*lines)++;
				}
			}
		}
	}
}

__global__ void accumulatorshoughtransform(uint8_t *im, uint32_t *accumulators, float *sin_table, float *cos_table, int width, int height,
	float hough_h, float center_x, float center_y){
		
		int j = (blockIdx.x * blockDim.x) + threadIdx.x, i = (blockIdx.y * blockDim.y) + threadIdx.y;
		int ii = threadIdx.x, jj = threadIdx.y;

		__shared__ float sh_sin[180 + PADDING];
		__shared__ float sh_cos[180 + PADDING];

		if((ii * BLOCKSZ) + jj < 180){
			sh_sin[(ii * BLOCKSZ) + jj] = sin_table[(ii * BLOCKSZ) + jj];
			sh_cos[(ii * BLOCKSZ) + jj] = cos_table[(ii * BLOCKSZ) + jj];
		}
		__syncthreads();

		int theta = 0;
		float rho = 0;

		if(i < height && j < width){
			if(im[ (i*width) + j] > 250){
				for(theta = 0; theta < 180; theta++) {  
					rho = ( ((float)j - center_x) * sh_cos[theta]) + (((float)i - center_y) * sh_sin[theta]);
					atomicAdd(&accumulators[(int)((round(rho + hough_h) * 180.0)) + theta], 1);
				} 
			}
		}
}


void line_asist_GPU(uint8_t *im, int height, int width,
	uint8_t *imEdge, float *NR, float *G, float *phi, float *Gx, float *Gy, uint8_t *pedge,
	float *sin_table, float *cos_table, 
	uint32_t *accum, int accu_height, int accu_width,
	int *x1, int *y1, int *x2, int *y2, int *nlines)
{
	int threshold;
	float hough_h = ((sqrt(2.0) * (float)(height>width?height:width)) / 2.0);
	float center_x = width/2.0; 
	float center_y = height/2.0;

	/**			canny		**/
	uint8_t *im_GPU, *imEdge_GPU;
	float   *NR_GPU, *G_GPU, *phi_GPU, *sin_table_GPU, *cos_table_GPU;
	uint32_t *accum_GPU;

	//Reserva de memoria
	hipMalloc((void**)&im_GPU,  sizeof(uint8_t) * width * height);
	hipMalloc((void**)&NR_GPU,  sizeof(float)   * width * height);
	hipMalloc((void**)&G_GPU,   sizeof(float)   * width * height);
	hipMalloc((void**)&phi_GPU, sizeof(float)   * width * height);
	hipMalloc((void**)&imEdge_GPU, sizeof(uint8_t) * width * height);

	dim3 dimBlock(BLOCKSZ,BLOCKSZ);
	dim3 dimGrid((width / dimBlock.x) + 1, (height / dimBlock.y) + 1);

	hipMemcpy(im_GPU,im, (sizeof(uint8_t) * width * height), hipMemcpyHostToDevice);
	NRcanny<<<dimGrid,dimBlock>>>(im_GPU, NR_GPU, height, width);
	hipDeviceSynchronize();

	Gcanny<<<dimGrid,dimBlock>>>(G_GPU, NR_GPU, phi_GPU, height, width);
	hipDeviceSynchronize();

	PEDGEcanny<<<dimGrid,dimBlock>>>(G_GPU, imEdge_GPU, phi_GPU, height, width, 1000.0f);
	hipDeviceSynchronize();

	/** 	hough transform 	**/
	//Reserva de memoria
	hipMalloc((void**)&accum_GPU,  sizeof(uint32_t) * accu_width * accu_height);
	hipMalloc((void**)&sin_table_GPU,  sizeof(float) * 180);
	hipMalloc((void**)&cos_table_GPU,  sizeof(float) * 180);

	hipMemset(accum_GPU, 0, (sizeof(uint32_t) * accu_width * accu_height));

	hipMemcpy(cos_table_GPU,cos_table, (sizeof(float) * 180), hipMemcpyHostToDevice);
	hipMemcpy(sin_table_GPU,sin_table, (sizeof(float) * 180), hipMemcpyHostToDevice);

	accumulatorshoughtransform<<<dimGrid,dimBlock>>>(imEdge_GPU, accum_GPU, sin_table_GPU, cos_table_GPU, width, height, hough_h, center_x, center_y);
	hipDeviceSynchronize();

	hipMemcpy(accum, accum_GPU, (sizeof(uint32_t) * accu_width * accu_height), hipMemcpyDeviceToHost);

	if (width>height) threshold = width/6;
	else threshold = height/6;

	getlines(threshold, accum, accu_width, accu_height, width, height, 
		sin_table, cos_table,
		x1, y1, x2, y2, nlines);
}
